#include "hip/hip_runtime.h"
#include <torch/types.h>
using namespace torch;

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>

// Cuda tensor accessor definitions
// restrict pointer traits piroritize speed over memory consumption
#define TensorAcc4R PackedTensorAccessor<scalar_t,4,RestrictPtrTraits,int32_t>
#define TensorAcc5R PackedTensorAccessor<scalar_t,5,RestrictPtrTraits,int32_t>
#define TensorAcc6R PackedTensorAccessor<scalar_t,6,RestrictPtrTraits,int32_t>
#define WITHIN_BOUNDS(x, y, H, W) (x >= 0 && x < H && y >= 0 && y < W)

#define THREADS_FORWARD 32
#define THREADS_BACKWARD 5


namespace {
template <typename scalar_t>
__global__ void correlation_cuda_forward_kernel(
    const TensorAcc4R rInput1,
    const TensorAcc4R rInput2,
    TensorAcc6R output,
    int kH, int kW,
    int patchH, int patchW,
    int padH, int padW,
    int dilation_patchH, int dilation_patchW,
    int dH, int dW) {

  const int iH = rInput1.size(2);
  const int iW = rInput1.size(3);
  const int C = rInput1.size(1);
  const int W = output.size(5);

  const int n = blockIdx.x;
  const int c = blockIdx.y;
  const int h = blockIdx.z;
  const int thread = threadIdx.x;

  const int start_i = -padH + h * dH;

  const int patchRadH = dilation_patchH * (patchH - 1) / 2;
  const int patchRadW = dilation_patchW * (patchW - 1) / 2;
  //printf(" %d ",patchRadH )
  //printf(" %d ",patchRadW )

  for (int w=thread; w<W; w += THREADS_FORWARD){
    int start_j = -padW + w * dW;
    int i1 = start_i;
    int j1 = start_j;
    for(int ph = 0; ph < patchH; ++ph){
      int ph_dilated = ph * dilation_patchH - patchRadH;
      for(int pw = 0; pw < patchW; ++pw){
        int pw_dilated = pw * dilation_patchW - patchRadW;
        int i2 = i1 + ph_dilated;
        if WITHIN_BOUNDS(i1, i2, iH, iH){
          int j2 = j1 + pw_dilated;
          if WITHIN_BOUNDS(j1, j2, iW, iW){
            scalar_t v1 = rInput1[n][c][i1][j1];
            scalar_t v2 = rInput2[n][c][i2][j2];
            output[n][c][pw][ph][h][w] += v1 * v2;  // avoid permute
          }
        }
      }
    }
  }
}


template <typename scalar_t>
__global__ void correlation_cuda_backward_kernel_input1(
    const TensorAcc6R gradOutput,
    const TensorAcc4R input2,
    TensorAcc4R gradInput1,
    int kH, int kW,
    int patchH, int patchW,
    int padH, int padW,
    int dilation_patchH, int dilation_patchW,
    int dH, int dW,
    int batch) {
  const int iH = input2.size(2);
  const int iW = input2.size(3);

  const int H = gradOutput.size(4);
  const int W = gradOutput.size(5);
  const int patchRadH = (patchH - 1) / 2;
  const int patchRadW = (patchW - 1) / 2;
  
  const int n = batch;
  const int c = blockIdx.x;
  const int h = blockIdx.y;
  const int w = blockIdx.z;
  const int ph_off = threadIdx.x;
  const int pw_off = threadIdx.y;

  const int h_2 = h + padH;
  const int w_2 = w + padW;
  const int start_i2 = h_2 / dH;
  const int start_j2 = w_2 / dW;
  /*we perform a module but since we have the quotient, we
  can cheat a bit*/
  const int h_off = h_2 - start_i2 * dH;
  const int w_off = w_2 - start_j2 * dW;

  __shared__ scalar_t prod_sum[THREADS_BACKWARD][THREADS_BACKWARD];
  prod_sum[ph_off][pw_off] = 0;

  for (int ph = ph_off; ph < patchH; ph += THREADS_BACKWARD) {
    int i1 = h + dilation_patchH * (ph - patchRadH);
    for (int pw = pw_off; pw < patchW; pw += THREADS_BACKWARD) {
      int j1 = w + dilation_patchW * (pw - patchRadW);
      if WITHIN_BOUNDS(i1, j1, iH, iW) {
        scalar_t val = input2[n][c][i1][j1];
        for(int tmp1 = h_off, i = 0; tmp1 < kH; tmp1 += dH, ++i) {
          int i2 = start_i2 - i;
          for(int tmp2 = w_off, j = 0; tmp2 < kW; tmp2 += dW, ++j) {
            int j2 = start_j2 - j;
            if WITHIN_BOUNDS(i2, j2, H, W) {
              prod_sum[ph_off][pw_off] += gradOutput[n][c][pw][ph][i2][j2] * val;
            }
          }
        }
      }
    }
  }

  __syncthreads();

  if (ph_off == 0 && pw_off == 0){
    scalar_t reduce_sum =0;
    for (int ph = 0; ph < THREADS_BACKWARD; ++ph){
      for (int pw = 0; pw < THREADS_BACKWARD; ++pw){
        reduce_sum += prod_sum[ph][pw];
      }
    }
    gradInput1[n][c][h][w] = reduce_sum;
  }
}


template <typename scalar_t>
__global__ void correlation_cuda_backward_kernel_input2(
    const TensorAcc6R gradOutput,
    const TensorAcc4R input1,
    TensorAcc4R gradInput2,
    int kH, int kW,
    int patchH, int patchW,
    int padH, int padW,
    int dilation_patchH, int dilation_patchW,
    int dH, int dW,
    int batch) {
  const int iH = input1.size(2);
  const int iW = input1.size(3);

  const int patchRadH = (patchH - 1) / 2;
  const int patchRadW = (patchW - 1) / 2;

  const int H = gradOutput.size(4);
  const int W = gradOutput.size(5);
  
  const int n = batch;
  const int c = blockIdx.x;
  const int h = blockIdx.y;
  const int w = blockIdx.z;
  const int ph_off = threadIdx.x;
  const int pw_off = threadIdx.y;

  __shared__ scalar_t prod_sum[THREADS_BACKWARD][THREADS_BACKWARD];
  prod_sum[ph_off][pw_off] = 0;

  for (int ph = ph_off; ph < patchH; ph += THREADS_BACKWARD) {
    int i1 = h - dilation_patchH * (ph - patchRadH);
    for (int pw = pw_off; pw < patchW; pw += THREADS_BACKWARD) {
      int j1 = w - dilation_patchW * (pw - patchRadW);
      if WITHIN_BOUNDS(i1, j1, iH, iW) {
        scalar_t val = input1[n][c][i1][j1];
        
        const int h_2 = i1 + padH;
        const int w_2 = j1 + padW;
        const int start_i2 = h_2 / dH;
        const int start_j2 = w_2 / dW;
        const int h_off = h_2 - start_i2 * dH;
        const int w_off = w_2 - start_j2 * dW;
        
        for(int tmp1 = h_off, i = 0; tmp1 < kH; tmp1 += dH, ++i) {
          int i2 = start_i2 - i;
          for(int tmp2 = w_off, j = 0; tmp2 < kW; tmp2 += dW, ++j) {
            int j2 = start_j2 - j;
            if WITHIN_BOUNDS(i2, j2, H, W) {
              prod_sum[ph_off][pw_off] += gradOutput[n][c][pw][ph][i2][j2] * val;
            }
          }
        }
      }
    }
  }

  __syncthreads();

  if (ph_off == 0 && pw_off == 0){
    scalar_t reduce_sum =0;
    for (int ph = 0; ph < THREADS_BACKWARD; ++ph){
      for (int pw = 0; pw < THREADS_BACKWARD; ++pw){
        reduce_sum += prod_sum[ph][pw];
      }
    }
    gradInput2[n][c][h][w] = reduce_sum;
  }
}
}


torch::Tensor correlation_cuda_forward(
    torch::Tensor input1,
    torch::Tensor input2,
    int kH, int kW,
    int patchH, int patchW,
    int padH, int padW,
    int dilation_patchH, int dilation_patchW,
    int dH, int dW) {

  const int batch_size = input1.size(0);
  const int nchannel = input1.size(1);
  const int iH = input1.size(2);
  const int iW = input1.size(3);

  const auto oH = (iH + 2 * padH - kH) / dH + 1;
  const auto oW = (iW + 2 * padW - kW) / dW + 1;

  auto output = at::zeros({batch_size, nchannel, patchW, patchH, oH, oW}, input1.options());
  auto trInput1 = input1;
  auto trInput2 = input2;
  //auto output = torch::zeros({batch_size, patchH, patchW, oH, oW}, input1.options());
  //auto trInput1 = input1.permute({0, 2, 3, 1}).contiguous();
  //auto trInput2 = input2.permute({0, 2, 3, 1}).contiguous();

  const int threads = THREADS_FORWARD;
  //const dim3 blocks(batch_size, oH, oW);
  const dim3 blocks(batch_size,nchannel,oH);
  //printf(" %d ",patchH );
  //printf(" %d ",patchW );

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input1.scalar_type(), "correlation_forward_cuda", ([&] {
    TensorAcc4R trInput1_acc  = trInput1.packed_accessor<scalar_t,4,RestrictPtrTraits,int32_t>();
    TensorAcc4R trInput2_acc = trInput2.packed_accessor<scalar_t,4,RestrictPtrTraits,int32_t>();
    TensorAcc6R output_acc = output.packed_accessor<scalar_t,6,RestrictPtrTraits,int32_t>();
    correlation_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
        trInput1_acc, trInput2_acc, output_acc,
        kH, kW, patchH, patchW, padH, padW,
        dilation_patchH, dilation_patchW, dH, dW);
  }));

  return output;
}

std::vector<torch::Tensor> correlation_cuda_backward(
    torch::Tensor input1,
    torch::Tensor input2,
    torch::Tensor gradOutput,
    int kH, int kW,
    int patchH, int patchW,
    int padH, int padW,
    int dilation_patchH, int dilation_patchW,
    int dH, int dW) {
  
  auto gradInput1 = torch::zeros_like(input1);
  auto gradInput2 = torch::zeros_like(input2);

  const int batch_size = input1.size(0);
  const int iH = input1.size(2);
  const int iW = input1.size(3);
  const int C = input1.size(1);

  const dim3 blocks(C, iH, iW);
  const dim3 threads(THREADS_BACKWARD, THREADS_BACKWARD);

  auto trInput1 = input1.contiguous();
  auto trInput2 = input2.contiguous();
  auto trgradOutput = gradOutput.contiguous();

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(trInput1.scalar_type(), "correlation_backward_cuda", ([&] {
    TensorAcc4R input1_acc = trInput1.packed_accessor<scalar_t,4,RestrictPtrTraits,int32_t>();
    TensorAcc4R input2_acc = trInput2.packed_accessor<scalar_t,4,RestrictPtrTraits,int32_t>();
    TensorAcc4R gradInput1_acc = gradInput1.packed_accessor<scalar_t,4,RestrictPtrTraits,int32_t>();
    TensorAcc4R gradInput2_acc = gradInput2.packed_accessor<scalar_t,4,RestrictPtrTraits,int32_t>();
    TensorAcc6R gradOutput_acc = trgradOutput.packed_accessor<scalar_t,6,RestrictPtrTraits,int32_t>();
//    TensorAcc5R gradOutput_acc = gradOutput.packed_accessor<scalar_t,5,RestrictPtrTraits,int32_t>();


    for (int n = 0; n < batch_size; ++n){
      correlation_cuda_backward_kernel_input1<scalar_t><<<blocks, threads>>>(
          gradOutput_acc, input2_acc, gradInput1_acc,
          kH, kW, patchH, patchW, padH, padW,
          dilation_patchH, dilation_patchW, dH, dW,
          n);
    }

    for (int n = 0; n < batch_size; ++n){
      correlation_cuda_backward_kernel_input2<scalar_t><<<blocks, threads>>>(
          gradOutput_acc, input1_acc, gradInput2_acc,
          kH, kW, patchH, patchW, padH, padW,
          dilation_patchH, dilation_patchW, dH, dW,
          n);
    }
  }
));

  return {gradInput1, gradInput2};
}
